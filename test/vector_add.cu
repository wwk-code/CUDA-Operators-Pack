
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// 核函数：向量加法
__global__ void vectorAdd(int *a, int *b, int *c, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n) {
        c[tid] = a[tid] + b[tid];  // 故意留一个越界bug（当tid == n时）
    }
}

int main() {
    int n = 1000;
    size_t size = n * sizeof(int);

    // 主机内存分配
    int *h_a = (int *)malloc(size);
    int *h_b = (int *)malloc(size);
    int *h_c = (int *)malloc(size);

    // 初始化数据
    for (int i = 0; i < n; i++) {
        h_a[i] = i;
        h_b[i] = i * 2;
    }

    // 设备内存分配
    int *d_a, *d_b, *d_c;
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    // 数据拷贝到设备
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    // 启动核函数（故意用n+1以触发bug）
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, n);

    // 拷贝结果回主机
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    // 验证结果（检查前10个元素）
    for (int i = 0; i < 10; i++) {
        printf("%d + %d = %d\n", h_a[i], h_b[i], h_c[i]);
    }

    // 释放内存
    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}