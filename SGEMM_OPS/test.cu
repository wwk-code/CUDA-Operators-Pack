#include "helper_string.h"
#include "hip/hip_runtime_api.h"

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <sys/time.h>
#include "utils.cuh"


#define MYSGEMM mysgemm_naive  // slect the kernel here


int main(int argc, char **argv) {
    if(argc != 2) {
        printf("Please select a kernel (range 0 - 11, here 0 is for NVIDIA cuBLAS).\n");
        exit(-1);
    }
    int SIZE[24];
    for(int i = 0; i<24;i++) SIZE[i] = (i+1) << 8;  // 256 - xxx
    int kernel_num = atoi(argv[1]);
    if(kernel_num < 0 || kernel_num > 11) {
        printf("Please enter a valid kernel number (0-11).\n");
        exit(-2);
    }
    int m,n,k,max_size;
    int n_count,N=10,upper_limit = 8;
    if(kernel_num<=4 && kernel_num!=0) upper_limit = 8;
    else upper_limit = (sizeof(SIZE)/sizeof(int));
    max_size = SIZE[upper_limit-1];   // 方阵中 行/列的最大元素数量
    FLOAT *A = NULL,*B = NULL, *C = NULL, *C_ref = NULL;
    FLOAT *dA = NULL,*dB = NULL, *dC = NULL, *dC_ref = NULL;
    FLOAT alpha = 1.0, beta = 0.;  // two arbitary input parameters
    float elapsed_time;
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipEvent_t begin,end;
    hipEventCreate(&begin);
    hipEventCreate(&end);
    A = (FLOAT*)malloc(sizeof(FLOAT) * max_size * max_size);
    B = (FLOAT*)malloc(sizeof(FLOAT) * max_size * max_size);
    C = (FLOAT*)malloc(sizeof(FLOAT) * max_size * max_size);
    C_ref = (FLOAT*)malloc(sizeof(FLOAT) * max_size * max_size);
    randomize_matrix(A,max_size*max_size);
    randomize_matrix(B,max_size*max_size);
    randomize_matrix(C,max_size*max_size);
    copy_matrix(C,C_ref,max_size*max_size);
    CUDA_CALLER(hipMalloc((void**) &dA,sizeof(FLOAT)*max_size*max_size));
    CUDA_CALLER(hipMalloc((void**) &dB,sizeof(FLOAT)*max_size*max_size));
    CUDA_CALLER(hipMalloc((void**) &dC,sizeof(FLOAT)*max_size*max_size));
    CUDA_CALLER(hipMalloc((void**) &dC_ref, sizeof(FLOAT)*max_size*max_size));
    CUDA_CALLER(hipMemcpy(dA,A,sizeof(FLOAT)*max_size*max_size,hipMemcpyHostToDevice));
    CUDA_CALLER(hipMemcpy(dB,B,sizeof(FLOAT)*max_size*max_size,hipMemcpyHostToDevice));
    CUDA_CALLER(hipMemcpy(dC,C,sizeof(FLOAT)*max_size*max_size,hipMemcpyHostToDevice));
    CUDA_CALLER(hipMemcpy(dC_ref,C_ref,sizeof(FLOAT)*max_size*max_size,hipMemcpyHostToDevice));

    for(int i_count = 0; i_count < upper_limit; i_count++) {
        m=n=k=SIZE[i_count];
        printf("\nM=N=K=%d:\n",m);
        if(kernel_num != 0) {
            hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,m,n,k,&alpha,dA,m,dB,k,&beta,dC_ref,m);
            test_kernel(kernel_num,m,n,k,alpha,dA,dB,beta,dC);
            hipDeviceSynchronize();
            hipMemcpy(C,dC,sizeof(FLOAT)*n*n,hipMemcpyDeviceToHost);
            hipMemcpy(C_ref,dC_ref,sizeof(FLOAT)*n*n,hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            if(!verify_matrix(C_ref,C,m*n)) {
                printf("Failed to pass the correctness verification against NVIDIA cuBLAS. Exited.\n");
                exit(-3);
            }
        }

        hipEventRecord(begin);
        if (kernel_num != 0){
            for (n_count=0;n_count<N;n_count++){
                test_kernel(kernel_num,m,n,k,alpha,dA,dB,beta,dC);
            }
        }else{
            for (n_count=0;n_count<N;n_count++){
                test_kernel(kernel_num,m,n,k,alpha,dA,dB,beta,dC, handle);
            }
        }
        hipEventRecord(end);
        hipEventSynchronize(begin);
        hipEventSynchronize(end);
        hipEventElapsedTime(&elapsed_time,begin,end);
        elapsed_time /= 1000.;
        printf("Average elasped time: %f second, performance: %f GFLOPS.\n", elapsed_time/N,2.*1e-9*N*m*n*k/elapsed_time);
        fflush(stdout);
        copy_matrix(C_ref,C,m*n); //sync C with cuBLAS to prepare for the next run，avoid of diff cumulation error
    }

    hipDeviceSynchronize();
    free(A);
    free(B);
    free(C);
    free(C_ref);
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    hipFree(dC_ref);
    hipDeviceSynchronize();

    return 0;

}





