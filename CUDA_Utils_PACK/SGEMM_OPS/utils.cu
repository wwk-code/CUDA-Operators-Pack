#include "hip/hip_runtime.h"
#include <ctime>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include "utils.cuh"
#include "kernels.cuh"
#include <helper_string.h> 
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#define FLOAT float
#define INT int

#define CEIL_DIV(m,n) (( m + n - 1) / n)

void print_matrix(const FLOAT *A, int m, int n){
    int i;
    printf("[");
    for (i = 0; i < m * n; i++){
        if ((i + 1) % n == 0) printf("%5.2f ", A[i]);
        else printf("%5.2f, ", A[i]);
        if ((i + 1) % n == 0){
            if (i + 1 < m * n) printf(";\n");
        }
    }
    printf("]\n");
}


void randomize_matrix(FLOAT *mat, int N) {
    srand(time(NULL));
    int i;
    for(i=0;i<N;i++) {
        FLOAT tmp = (float)(rand() % 5) + 0.01 * (rand() % 5);
        tmp = (rand() % 2 == 0) ? tmp : tmp * (-1.);
        mat[i] = tmp;
    }
}


double get_sec() {
    struct timeval time;
    gettimeofday(&time, NULL);
    return (time.tv_sec + 1e-6 * time.tv_usec);  // 1e-6 = 10^(-6), tv_usec为微秒数，tv_sec为自1970年来过了多少秒
}


bool verify_matrix(FLOAT *mat1, FLOAT *mat2, int n) {
    double diff = 0.0;
    int i ;
    for(i = 0; mat1 + i && mat2+i && i<n; i++) {
        diff = fabs((double)mat1[i] - (double)mat2[i]);
        if(diff > 1e-2) {
            printf("error. %5.2f,%5.2f,%d\n", mat1[i],mat2[i],i);
            return false;
        }
    }
    return true;
}


// 自定义的矩阵拷贝函数
void copy_matrix(FLOAT *src, FLOAT *dest, int n){
    int i;
    for (i = 0; src + i && dest + i && i < n; i++) *(dest + i) = *(src + i);
    if (i != n) printf("copy failed at %d while there are %d elements in total.\n", i, n);
}


void test_cublas(hipblasHandle_t handle, INT M, INT N, INT K, FLOAT alpha, FLOAT *A, FLOAT *B, FLOAT beta, FLOAT *C){
    hipDeviceSynchronize();
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, A, M, B, K, &beta, C, M);
    hipDeviceSynchronize();
}


void test_mysgemm_v1(INT M, INT N, INT K,FLOAT alpha, FLOAT *A, FLOAT *B, FLOAT beta, FLOAT *C) {
    hipDeviceSynchronize();
    dim3 blockDim(32,32);
    dim3 gridDim(CEIL_DIV(M,32),CEIL_DIV(N,32));
    mysgemm_v1<<<gridDim,blockDim>>>(M,N,K,alpha,A,B,beta,C);
    hipDeviceSynchronize();
}


void test_mysgemm_v2(INT M, INT N, INT K,FLOAT alpha, FLOAT *A, FLOAT *B, FLOAT beta, FLOAT *C) {
    hipDeviceSynchronize();
    dim3 blockDim(32,32);
    dim3 gridDim(CEIL_DIV(M,32),CEIL_DIV(N,32));
    mysgemm_v2<<<gridDim,blockDim>>>(M,N,K,alpha,A,B,beta,C);
    hipDeviceSynchronize();
}


void test_mysgemm_v3(INT M, INT N, INT K, FLOAT alpha, FLOAT *A, FLOAT *B, FLOAT beta, FLOAT *C){
    hipDeviceSynchronize();
    dim3 blockDim(1024);
    dim3 gridDim(CEIL_DIV(M,32),CEIL_DIV(N,32));
    mysgemm_v3<<<gridDim, blockDim>>>(M,N,K,alpha,A,B,beta,C);
    hipDeviceSynchronize();
}

void test_mysgemm_v4(INT M, INT N, INT K, FLOAT alpha, FLOAT *A, FLOAT *B, FLOAT beta, FLOAT *C){
    hipDeviceSynchronize();
    dim3 blockDim(1024);
    dim3 gridDim(CEIL_DIV(M,32),CEIL_DIV(N,32));
    mysgemm_v4<<<gridDim, blockDim>>>(M,N,K,alpha,A,B,beta,C);
    hipDeviceSynchronize();
}



void test_mysgemm_v5(INT M, INT N, INT K, FLOAT alpha, FLOAT *A, FLOAT *B, FLOAT beta, FLOAT *C){
    hipDeviceSynchronize();
    dim3 blockDim(256);
    dim3 gridDim(CEIL_DIV(M,32),CEIL_DIV(N,32));
    mysgemm_v5<<<gridDim, blockDim>>>(M,N,K,alpha,A,B,beta,C);
    hipDeviceSynchronize();
}


void test_mysgemm_v6(INT M, INT N, INT K, FLOAT alpha, FLOAT *A, FLOAT *B, FLOAT beta, FLOAT *C){
    hipDeviceSynchronize();
    dim3 blockDim(256);
    dim3 gridDim(CEIL_DIV(M,32),CEIL_DIV(N,32));
    mysgemm_v6<<<gridDim, blockDim>>>(M,N,K,alpha,A,B,beta,C);
    hipDeviceSynchronize();
}


void test_mysgemm_v7(INT M, INT N, INT K, FLOAT alpha, FLOAT *A, FLOAT *B, FLOAT beta, FLOAT *C){
    hipDeviceSynchronize();
    dim3 blockDim(256);
    dim3 gridDim(CEIL_DIV(M,64),CEIL_DIV(N,64));
    mysgemm_v7<<<gridDim, blockDim>>>(M,N,K,alpha,A,B,beta,C);
    hipDeviceSynchronize();
}


void test_mysgemm_v8(INT M, INT N, INT K, FLOAT alpha, FLOAT *A, FLOAT *B, FLOAT beta, FLOAT *C){
    hipDeviceSynchronize();
    dim3 blockDim(256);
    dim3 gridDim(CEIL_DIV(M,128),CEIL_DIV(N,128));
    mysgemm_v8<<<gridDim, blockDim>>>(M,N,K,alpha,A,B,beta,C);
    hipDeviceSynchronize();
}



void test_kernel(int kernel_num,INT m,INT n,INT k,FLOAT alpha,FLOAT *A,FLOAT *B,FLOAT beta,FLOAT *C, hipblasHandle_t handle){
    switch (kernel_num){
        case 0: test_cublas(handle, m,n,k,alpha,A,B,beta,C); break;
        case 1: test_mysgemm_v1(m,n,k,alpha,A,B,beta,C); break;
        case 2: test_mysgemm_v2(m,n,k,alpha,A,B,beta,C); break;
        case 3: test_mysgemm_v3(m,n,k,alpha,A,B,beta,C); break;
        case 4: test_mysgemm_v4(m,n,k,alpha,A,B,beta,C); break;
        case 5: test_mysgemm_v5(m,n,k,alpha,A,B,beta,C); break;
        case 6: test_mysgemm_v6(m,n,k,alpha,A,B,beta,C); break;
        case 7: test_mysgemm_v7(m,n,k,alpha,A,B,beta,C); break;
        case 8: test_mysgemm_v8(m,n,k,alpha,A,B,beta,C); break;
        // case 9: test_mysgemm_v9(m,n,k,alpha,A,B,beta,C); break;
        // case 10: test_mysgemm_v10(m,n,k,alpha,A,B,beta,C); break;
        // case 11: test_mysgemm_v11(m,n,k,alpha,A,B,beta,C); break;
        default: break;
    }
}











