#include <iostream>
#include <hip/hip_runtime.h>



// 使用高斯消元法求矩阵的逆
__global__ void matReverse_kernel(float *A,float *I, int N) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx >= N) return; 

    for(int i = 0;i < N; i++) {
        float diagValue = A[i * N + i];
        if(idx == i) {
            for(int j = 0;j < N; j++) {
                A[i * N + j] = i == j ? 1.0f : 0.0f;
                I[i * N + j] = i == j ? 1.0f : 0.0f;
            }
        }
        __syncthreads();
        if(idx != i) {
            float rowValue = A[idx * N + i]; 
            for(int j = 0;j < N; j++) {
                A[idx * N + j] -= A[i * N + j] * rowValue;
            }
        }
    }
}


void matrixInverse(float *A, float *I, int N) {
    float *d_A, *d_I;
    int size = N * N * sizeof(float);
    hipMalloc((void**)&d_A,size);
    hipMalloc((void**)&d_I,size);
    hipMemcpy(d_A,A,size,hipMemcpyHostToDevice);
    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;
    matReverse_kernel<<<gridSize,blockSize>>>(d_A,d_I,N);
    hipMemcpy(I,d_I,size,hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_I);
}

int main() {
    const int N = 3;
    float A[N * N] = {1, 3, 5,-1, 2, 3,4, 0, 6};
    float I[N * N];

    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            std::cout << A[i * N + j] << " ";
        }
        std::cout << std::endl;
    }

    matrixInverse(A, I, N);

    std::cout << "Inverse matrix:" << std::endl;
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            std::cout << I[i * N + j] << " ";
        }
        std::cout << std::endl;
    }

    return 0;
}

